#include <hip/hip_runtime.h>
#include <hipblaslt.h>
#include <hip/hip_bf16.h>
#include <random>
#include <vector>
#include <chrono>
#include <iostream>
#include <iomanip>

void check(hipError_t error) {
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void checkCublas(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS error: " << status << std::endl;
        exit(EXIT_FAILURE);
    }
}

double benchmark_matmul(int matrix_size) {
    std::cout << "\nBenchmarking size: " << matrix_size << "x" << matrix_size << std::endl;
    
    // Initialize dimensions
    const int m = matrix_size;
    const int n = matrix_size;
    const int k = matrix_size;
    
    // Allocate host memory
    std::random_device rd;
    std::mt19937 gen(rd());
    std::normal_distribution<float> dist(0.0f, std::sqrt(std::sqrt(1.0f/k)));
    std::vector<__hip_bfloat16> h_A(m * k);
    std::vector<__hip_bfloat16> h_B(k * n);
    for(int i = 0; i < m * k; i++) {
        h_A[i] = __float2bfloat16(dist(gen));
    }
    for(int i = 0; i < k * n; i++) {
        h_B[i] = __float2bfloat16(dist(gen));
    }
    std::vector<__hip_bfloat16> h_C(m * n, __hip_bfloat16(0.0f));
    
    // Allocate device memory
    __hip_bfloat16 *d_A, *d_B, *d_C;
    uint8_t *workspace;
    check(hipMalloc(&workspace, 32 * 1024 * 1024));
    check(hipMalloc(&d_A, m * k * sizeof(__hip_bfloat16)));
    check(hipMalloc(&d_B, k * n * sizeof(__hip_bfloat16)));
    check(hipMalloc(&d_C, m * n * sizeof(__hip_bfloat16)));
    
    // Copy data to device
    check(hipMemcpy(d_A, h_A.data(), m * k * sizeof(__hip_bfloat16), hipMemcpyHostToDevice));
    check(hipMemcpy(d_B, h_B.data(), k * n * sizeof(__hip_bfloat16), hipMemcpyHostToDevice));
    
    // Initialize cuBLASLt
    hipblasLtHandle_t handle;
    checkCublas(hipblasLtCreate(&handle));
    
    // Configure matrix descriptors
    hipblasLtMatrixLayout_t matA, matB, matC; // apparently column major is strongly preferred (???)
    checkCublas(hipblasLtMatrixLayoutCreate(&matA, HIP_R_16BF, m, k, m));
    checkCublas(hipblasLtMatrixLayoutCreate(&matB, HIP_R_16BF, k, n, k));
    checkCublas(hipblasLtMatrixLayoutCreate(&matC, HIP_R_16BF, m, n, m));
    
    // Configure matrix multiplication descriptor
    hipblasLtMatmulDesc_t matmulDesc;
    checkCublas(hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32F_FAST_16BF, HIP_R_32F));
    
    // Set matrix operation parameters
    const float alpha = 1.0f;
    const float beta = 0.0f;

    size_t workspaceSize = 32 * 1024 * 1024;
    
    // Create preference descriptor
    hipblasLtMatmulPreference_t preference;
    checkCublas(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublas(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));
    
    // Query the best algorithm
    int returnedResults = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult;
    checkCublas(hipblasLtMatmulAlgoGetHeuristic(
        handle, matmulDesc, matA, matB, matC, matC, preference, 1, &heuristicResult, &returnedResults
    ));
    std::cout << "Returned results: " << returnedResults << std::endl;

    // Warmup iterations
    for (int i = 0; i < 10; i++) {
        checkCublas(hipblasLtMatmul(
            handle,
            matmulDesc,
            &alpha,
            d_A, matA,
            d_B, matB,
            &beta,
            d_C, matC,
            d_C, matC,
            &heuristicResult.algo,
            workspace, workspaceSize,
            0
        ));
    }
    
    // Synchronize before timing
    check(hipDeviceSynchronize());
    
    // Timing iterations
    const int NUM_ITERATIONS = 10;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < NUM_ITERATIONS; i++) {
        checkCublas(hipblasLtMatmul(
            handle,
            matmulDesc,
            &alpha,
            d_A, matA,
            d_B, matB,
            &beta,
            d_C, matC,
            d_C, matC,
            &heuristicResult.algo,
            workspace, workspaceSize,
            0
        ));
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float avg_time = milliseconds / NUM_ITERATIONS;
    
    // Calculate TFLOPS
    double flops = 2.0 * m * n * k; // multiply-add counts as 2 operations
    double tflops = (flops / (avg_time * 1e-3)) / 1e12;
    
    std::cout << "Average time: " << std::fixed << std::setprecision(2) << avg_time 
              << " ms, Performance: " << std::setprecision(2) << tflops << " TFLOPS" << std::endl;
    
    // Verify correctness on random indices
    // Copy matrices back to host
    check(hipMemcpy(h_C.data(), d_C, m * n * sizeof(__hip_bfloat16), hipMemcpyDeviceToHost));

    // Seed random number generator
    std::uniform_int_distribution<> dis_m(0, m-1);
    std::uniform_int_distribution<> dis_n(0, n-1);

    // Check 50 random positions
    for (int i = 0; i < 50; i++) {
        int row = dis_m(gen);
        int col = dis_n(gen);
        
        // Calculate expected value
        float expected = 0.0f;  // Use float for intermediate computation
        for (int j = 0; j < k; j++) {
            expected += __bfloat162float(h_A[j * m + row]) * __bfloat162float(h_B[col * k + j]);
        }
        expected = alpha * expected + beta * __bfloat162float(h_C[row * n + col]);
        
        // Get actual value and convert to float for comparison
        float actual = __bfloat162float(h_C[col * m + row]);
        
        // Compare with larger tolerance due to bf16 precision
        float rel_error = std::abs(actual - expected) / std::abs(expected);
        if (rel_error > 0.01) {  // Increased tolerance for bf16
            std::cout << "Verification failed at position [" << row << "," << col << "]" << std::endl;
            std::cout << "Expected: " << expected << ", Got: " << actual << ", Relative Error: " << rel_error << std::endl;
        }
        else {
            if(i < 5) {
                std::cout << "Verification passed at position [" << row << "," << col << "]" << " with values " << expected << " and " << actual << std::endl;
            }
        }
    }

    // Cleanup
    hipblasLtMatmulPreferenceDestroy(preference);
    hipblasLtMatrixLayoutDestroy(matA);
    hipblasLtMatrixLayoutDestroy(matB);
    hipblasLtMatrixLayoutDestroy(matC);
    hipblasLtMatmulDescDestroy(matmulDesc);
    hipblasLtDestroy(handle);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return tflops;
}

int main() {
    // Initialize CUDA
    int device = 0;
    hipSetDevice(device);
    
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);
    std::cout << "Running on GPU: " << deviceProp.name << std::endl;
    
    // Matrix sizes to benchmark
    std::vector<int> sizes = {1024, 2048, 4096, 8192, 16384};
    
    // Run benchmarks
    for (int size : sizes) {
        benchmark_matmul(size);
    }
    
    return 0;
}