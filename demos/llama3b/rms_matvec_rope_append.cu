#include "hip/hip_runtime.h"
#include "llama.cuh"
#include "utils.cuh"
#include "matvec_pipeline.cuh"

using namespace kittens;
using namespace megakernel;

using globals = llama_3b_globals;

template <typename Config, typename Globals> struct rms_qkv_rope_append {
    static constexpr int opcode =
        OPCODE_RMS_QKV_MatVecRopeAppend; // Op index within the layer --
                                         // controls which barrier to listen to.

    static constexpr int K_BLK_START = 3072 / Globals::matvec_block_size;
    static constexpr int V_BLK_START = 4096 / Globals::matvec_block_size;
    static constexpr int EXPECTED_ARRIVAL_COUNT = 384;

    using rope_t = kittens::sv_fl<Globals::head_dim>;

    __device__ static inline uint8_t *get_rope_cos_ptr(megakernel::state<Config> &s) {
        return (uint8_t *)s.scratch() + Config::SCRATCH_BYTES - 1024;
    }
    __device__ static inline uint8_t *get_rope_sin_ptr(megakernel::state<Config> &s) {
        return (uint8_t *)s.scratch() + Config::SCRATCH_BYTES - 512;
    }
    __device__ static inline rope_t &get_rope_cos(megakernel::state<Config> &s) {
        return *reinterpret_cast<rope_t *>(get_rope_cos_ptr(s));
    }
    __device__ static inline rope_t &get_rope_sin(megakernel::state<Config> &s) {
        return *reinterpret_cast<rope_t *>(get_rope_sin_ptr(s));
    }

    struct parsed_instruction {
        int layer_idx, start_block_idx, end_block_idx, iters;
        __device__ inline parsed_instruction(
            typename Config::instruction_t &instruction) {
            layer_idx = instruction[1];       // in units of 1
            start_block_idx = instruction[2]; // in units of 16 elements
            end_block_idx = instruction[3];   // in units of 16 elements
            iters = end_block_idx - start_block_idx;
        }
        __device__ inline parsed_instruction(megakernel::state<Config> &s)
            : parsed_instruction(s.instruction()) {}
    };

    struct pipeline_specifics {

        static __device__ inline void gmem_wait(const Globals &g,
                                                megakernel::state<Config> &s) {
            parsed_instruction inst{s};
            if (inst.layer_idx > 0) {
                while (
                    *(volatile int *)&g.Bar[{inst.layer_idx - 1,
                                             OPCODE_DownProjResidual - 1, 0}] <
                    EXPECTED_ARRIVAL_COUNT) {
                    // if (kittens::laneid() == 0){
                    //     printf("Waiting for layer %d to finish, current count: %d / %d\n",
                    //            inst.layer_idx - 1,
                    //            *(volatile int *)&g.Bar[{inst.layer_idx - 1,
                    //                                     OPCODE_DownProjResidual - 1, 0}],
                    //            EXPECTED_ARRIVAL_COUNT);
                    // }
                    __nanosleep(Config::GMEM_SPIN_LOOP_SLEEP_NANOS);
                }
            }
        }

        static __device__ inline void
        load_iter(megakernel::state<Config> &s, const globals &g, parsed_instruction &inst,
                  int iter, int col_idx, kittens::st_bf<16, 512> &weight_chunk,
                  kittens::semaphore &sem) {
            auto block_idx = inst.start_block_idx + iter;
            kittens::tma::load_async<dim::ROW, cache_policy::EVICT_FIRST>(
                weight_chunk, g.qkv_weights,
                {inst.layer_idx, block_idx, col_idx}, sem);
        }

        static __device__ inline void store(megakernel::state<Config> &s, const Globals &g,
                                            parsed_instruction &inst,
                                            int output_idx, int output_stage) {
            int block_idx = inst.start_block_idx + output_idx;

            // apply rope

            // even for V, we need to cast from float to bf16
            uint8_t *output_scratch_start =
                pipeline::get_output_start(s, output_stage);

            // kittens::sv_fl<16> &qkv_proj_smem = *reinterpret_cast<kittens::sv_fl<16>
            // *>(output_scratch_start);
            kittens::sv_bf<16> &qkv_proj_smem_bf =
                *reinterpret_cast<kittens::sv_bf<16> *>(output_scratch_start);

            kittens::rv_fl<16> qkv_proj, rope_cos, rope_sin;

            matvec_reduce<Config, kittens::sv_fl<16>, kittens::rv_fl<16>,
                          pipeline::SCRATCH_BYTES_PER_WARP>(
                output_scratch_start, qkv_proj);

            kittens::wait(rope_arrived(s), 0);

            auto head_chunk = block_idx % 8;

            kittens::sv_fl<16> &rope_cos_sv = *reinterpret_cast<kittens::sv_fl<16> *>(
                get_rope_cos_ptr(s) + head_chunk * 64);
            kittens::sv_fl<16> &rope_sin_sv = *reinterpret_cast<kittens::sv_fl<16> *>(
                get_rope_sin_ptr(s) + head_chunk * 64);

            kittens::warp::load(rope_cos, rope_cos_sv);
            kittens::warp::load(rope_sin, rope_sin_sv);

            // if (s.instruction()[1] == 0 && block_idx < 8)
            //     printf("block_idx %d, rope_cos[%d]=%f\n", block_idx, kittens::laneid(), __bfloat162float(rope_cos[0][0]));
            // if (s.instruction()[1] == 0 && kittens::laneid() < 16)
            //     printf("block_idx %d, qkv_proj[%d]=%f\n", block_idx, kittens::laneid(), __bfloat162float(qkv_proj[0][0]));

            if (block_idx < V_BLK_START) { // only Q & K need RoPE

                // Fetch the neighbor values
                int mod = (kittens::laneid() & 0b1) ? -1 : 1; // 1 for even, -1 for odd
                kittens::warp::sync();
                float pair_val =
                    __shfl_sync(MASK_ALL, qkv_proj[0][0], kittens::laneid() + mod);

                // Compute RoPE in-place
                if (kittens::laneid() < 16) {
                    // will clean this up later
                    qkv_proj[0][0] =
                        float(qkv_proj[0][0]) * rope_cos[0][0] +
                        float(-1 * mod) * float(pair_val) * rope_sin[0][0];
                }
            }

            // if (s.instruction()[1] == 0 && kittens::laneid() < 16)
            //     printf("block_idx %d, qkv_proj[%d]=%f\n", block_idx, kittens::laneid(), __bfloat162float(qkv_proj[0][0]));

            kittens::warp::sync();
            kittens::warp::store(qkv_proj_smem_bf, qkv_proj);
            // for (int i = 0; i < qkv_proj_smem_bf.length; i++) {
            //     if (isnan(__bfloat162float(qkv_proj_smem_bf[i]))) {
            //         printf("NaN detected in qkv_proj_smem_bf at lane %d, "
            //                "block_idx %d, head_chunk %d, output_stage %d, output_idx %d\n",
            //                kittens::laneid(), block_idx, head_chunk, output_stage, output_idx);
            //     }
            // }
                
            kittens::warp::sync();

            if (kittens::laneid() == 0) {
                // printf("inst.start_block_idx %d, head_chunk %d, "
                //     "output_stage %d, output_idx %d, block_idx %d\n",
                //     inst.start_block_idx, head_chunk, output_stage, output_idx, block_idx);
                if (block_idx < K_BLK_START) { // Q
                    kittens::tma::store_async<cache_policy::EVICT_LAST>(
                        g.q_post_rope, qkv_proj_smem_bf, {0, 0, 0, block_idx});
                } else if (block_idx < V_BLK_START) { // K
                    int base_index =
                        (block_idx - K_BLK_START) * Globals::matvec_block_size;
                    int head_idx = base_index / Globals::head_dim;
                    int dim_idx = (base_index % Globals::head_dim) /
                                  Globals::matvec_block_size;
                    kittens::tma::store_async<cache_policy::EVICT_LAST>(
                        g.k_cache, qkv_proj_smem_bf,
                        {inst.layer_idx, static_cast<int>(g.pos_id), head_idx,
                         dim_idx});
                } else { // V
                    int base_index =
                        (block_idx - V_BLK_START) * Globals::matvec_block_size;
                    int head_idx = base_index / Globals::head_dim;
                    int dim_idx = (base_index % Globals::head_dim) /
                                  Globals::matvec_block_size;
                    kittens::tma::store_async<cache_policy::EVICT_LAST>(
                        g.v_cache, qkv_proj_smem_bf,
                        {inst.layer_idx, static_cast<int>(g.pos_id), head_idx,
                         dim_idx});
                }

                // printf("block_idx %d, qkv_proj_smem_bf[0]=%f, qkv_proj_smem_bf[1]=%f, qkv_proj_smem_bf[2]=%f, qkv_proj_smem_bf[3]=%f, "
                //        "qkv_proj_smem_bf[4]=%f, qkv_proj_smem_bf[5]=%f, qkv_proj_smem_bf[6]=%f, qkv_proj_smem_bf[7]=%f, "
                //        "qkv_proj_smem_bf[8]=%f, qkv_proj_smem_bf[9]=%f, qkv_proj_smem_bf[10]=%f, qkv_proj_smem_bf[11]=%f, "
                //        "qkv_proj_smem_bf[12]=%f, qkv_proj_smem_bf[13]=%f, qkv_proj_smem_bf[14]=%f, qkv_proj_smem_bf[15]=%f\n",
                //     block_idx,
                //     __bfloat162float(qkv_proj_smem_bf.data[0]), __bfloat162float(qkv_proj_smem_bf.data[1]), __bfloat162float(qkv_proj_smem_bf.data[2]), __bfloat162float(qkv_proj_smem_bf.data[3]),
                //     __bfloat162float(qkv_proj_smem_bf.data[4]), __bfloat162float(qkv_proj_smem_bf.data[5]), __bfloat162float(qkv_proj_smem_bf.data[6]), __bfloat162float(qkv_proj_smem_bf.data[7]),
                //     __bfloat162float(qkv_proj_smem_bf.data[8]), __bfloat162float(qkv_proj_smem_bf.data[9]), __bfloat162float(qkv_proj_smem_bf.data[10]), __bfloat162float(qkv_proj_smem_bf.data[11]),
                //     __bfloat162float(qkv_proj_smem_bf.data[12]), __bfloat162float(qkv_proj_smem_bf.data[13]), __bfloat162float(qkv_proj_smem_bf.data[14]), __bfloat162float(qkv_proj_smem_bf.data[15])
                // );                
                s.record(megakernel::TEVENT_AT_GMEM_STORE);

                kittens::tma::store_async_wait(); // not just read wait! full wait! must
                                         // be visible in global!
                // asm volatile("fence.acq_rel.gpu;\n"); // possible we need sc
                // here but I don't think so.

                atomicAdd(&g.Bar[{inst.layer_idx, opcode - 1, block_idx / 8}],
                          1);
                // printf(
                //     "block %d, opcode %d, Bar[%d, %d, %d] = %d\n",
                //     block_idx, opcode, inst.layer_idx, opcode - 1,
                //     block_idx / 8,
                //     g.Bar[{inst.layer_idx, opcode - 1, block_idx / 8}]);
                s.record(megakernel::TEVENT_DONE_GMEM_STORE);
            }

            kittens::warp::sync();
        }
    };

    using pipeline =
        rms_matvec_pipeline<Config, Globals, parsed_instruction,
                            pipeline_specifics, &Globals::hidden_states,
                            &Globals::attn_norm_weights>;

    __device__ static inline kittens::semaphore &rope_arrived(megakernel::state<Config> &s) {
        return s.semaphores()[pipeline::SEM_COUNT];
    }

    struct controller {
        static __device__ int
        release_lid(const Globals &g,
                    typename Config::instruction_t &instruction, int &query) {
            return pipeline::release_lid(g, instruction, query);
        }
        static __device__ int init_semaphores(const Globals &g,
                                              megakernel::state<Config> &s) {
            pipeline::init_semaphores(s);
            init_semaphore(rope_arrived(s), 1);
            return pipeline::SEM_COUNT + 1;
        }
    };
    struct loader {
        static __device__ void run(const Globals &g, megakernel::state<Config> &s) {
            if (kittens::laneid() == 0) {
                auto &rope_cos = get_rope_cos(s);
                auto &rope_sin = get_rope_sin(s);

                auto &sem = rope_arrived(s);
                kittens::tma::expect(sem, rope_cos, rope_sin);

                kittens::tma::load_async<cache_policy::EVICT_LAST>(
                    rope_cos, g.rope_cos, {0, 0, static_cast<int>(g.pos_id), 0},
                    sem);
                kittens::tma::load_async<cache_policy::EVICT_LAST>(
                    rope_sin, g.rope_sin, {0, 0, static_cast<int>(g.pos_id), 0},
                    sem);
            }

            parsed_instruction inst{s};
            pipeline::loader_loop(s, g, inst.layer_idx);
        }
    };
    struct launcher {
        static __device__ void run(const Globals &g, megakernel::state<Config> &s) {

            parsed_instruction inst{s};
            pipeline::launcher_loop(s, g);
        }
    };
    struct consumer {
        static __device__ void run(const Globals &g, megakernel::state<Config> &s) {
            pipeline::consumer_loop(s, g);
        }
    };
    struct storer {
        // Uses 4 full pages for outputs.
        static __device__ void run(const Globals &g, megakernel::state<Config> &s) {
            pipeline::storer_loop(s, g);
        }
    };
};
