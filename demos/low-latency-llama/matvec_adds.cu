#include "hip/hip_runtime.h"
#pragma once

#include "llama.cuh"
#include "utils.cuh"
#include "matvec_pipeline.cuh"

using namespace kittens;
using namespace megakernel;

template <int _EXPECTED_ARRIVAL_COUNT, auto WeightsPtr,
          auto InputActivationsPtr, auto OutputActivationsPtr, int _opcode,
          int _prev_opcode = 0,
          typename Config = default_config,
          typename Globals = llama_1b_globals>

struct MatVecAddOp {
    static constexpr int opcode = _opcode;
    static constexpr int prev_opcode = _prev_opcode;
    static constexpr int EXPECTED_ARRIVAL_COUNT = _EXPECTED_ARRIVAL_COUNT;

    struct parsed_instruction {
        int layer, start_block_idx, end_block_idx, reduction_block_idx,
            start_reduction_col, iters;
        __device__ inline parsed_instruction(
            typename Config::instruction_t &instruction) {
            layer = instruction[1]; // in units of 1
            start_block_idx =
                instruction[2]; // in units of 1 (0, 16, 32, ..., 2032)
            end_block_idx =
                instruction[3]; // in units of 1 (0, 16, 32, ..., 2032)
            reduction_block_idx = instruction[4]; // in units of hidden_dim=2048
                                                  // (0, 2048, 4096, 6144)
            start_reduction_col = reduction_block_idx * Globals::hidden_dim;
            iters = end_block_idx - start_block_idx;
        }
        __device__ inline parsed_instruction(megakernel::state<Config> &s)
            : parsed_instruction(s.instruction()) {}
    };

    struct pipeline_specifics {

        static __device__ inline void
        load_iter(megakernel::state<Config> &s, const globals &g, parsed_instruction &inst,
                  int iter, int col_idx, kittens::st_bf<16, 512> &weight_chunk,
                  kittens::semaphore &sem) {
            kittens::tma::load_async<dim::ROW, cache_policy::EVICT_FIRST>(
                weight_chunk, g.*WeightsPtr,
                coord<>{inst.layer,
                        (inst.start_block_idx + iter) *
                            Globals::matvec_block_size,
                        inst.start_reduction_col + 512 * col_idx},
                sem);
        }

        static __device__ inline void store(megakernel::state<Config> &s, const globals &g,
                                            parsed_instruction &inst,
                                            int output_idx, int output_stage) {

            int block_idx = inst.start_block_idx + output_idx;

            uint8_t *output_scratch_start =
                pipeline::get_output_start(s, output_stage);
            kittens::sv_bf<16> &output_smem_bf =
                *reinterpret_cast<kittens::sv_bf<16> *>(output_scratch_start);

            kittens::rv_fl<16> output_rv;
            matvec_reduce<Config, kittens::sv_fl<16>, kittens::rv_fl<16>,
                          pipeline::SCRATCH_BYTES_PER_WARP>(
                output_scratch_start, output_rv);

            kittens::warp::sync();
            kittens::warp::store(output_smem_bf, output_rv);
            kittens::warp::sync();

            if (kittens::warp::laneid() == 0) {
                auto &OutputActivations =
                    g.*OutputActivationsPtr; // object in global memory
                kittens::tma::store_add_async<cache_policy::EVICT_LAST>(
                    OutputActivations, output_smem_bf, {block_idx});
                kittens::tma::store_async_read_wait();
            }

            kittens::warp::sync();
        }
    };
    using pipeline = matvec_pipeline<Config, Globals, parsed_instruction,
                                     pipeline_specifics>;

    struct controller {
        static __device__ int
        release_lid(const Globals &g,
                    typename Config::instruction_t &instruction, int &query) {
            return pipeline::release_lid(g, instruction, query);
        }

        static __device__ int init_semaphores(const Globals &g,
                                              megakernel::state<Config> &s) {
            return pipeline::init_semaphores(s);
        }
    };
    struct loader {
        static __device__ void run(const Globals &g, megakernel::state<Config> &s) {
            pipeline::loader_loop(s, g);
        }
    };
    struct launcher {
        static __device__ void run(const globals &g, megakernel::state<Config> &s) {
            if (kittens::laneid() == 0) {
#ifdef KITTENS_BLACKWELL
                s.wait_tensor_ready();
                arrive(s.tensor_finished, Config::NUM_CONSUMER_WARPS);
#endif
            }
        }
    };
    struct consumer {
        static __device__ void run(const Globals &g, megakernel::state<Config> &s) {

            using sv_t = kittens::sv_bf<pipeline::REDUCTION_DIM_PER_WARP>;
            using rv_t = kittens::rv_fl<pipeline::REDUCTION_DIM_PER_WARP>;
            parsed_instruction inst{s};

            if (kittens::laneid() == 0 && kittens::warpid() == 0) {

                int activation_page = pipeline::get_activation_page(s);
                s.wait_page_ready(activation_page);

                s.record(megakernel::TEVENT_AT_GMEM_WAIT);
                while (*(volatile int *)&g.Bar[{inst.layer, prev_opcode - 1,
                                                inst.reduction_block_idx}] <
                       EXPECTED_ARRIVAL_COUNT) {
                    __nanosleep(Config::GMEM_SPIN_LOOP_SLEEP_NANOS);
                }
                s.record(megakernel::TEVENT_DONE_GMEM_WAIT);

                auto &activations = pipeline::get_activations(s);
                auto &InputActivations =
                    g.*InputActivationsPtr; // object in global memory
            }
            group<Config::NUM_CONSUMER_WARPS>::sync(4);

            sv_t &activations_smem = reinterpret_cast<sv_t *>(
                &pipeline::get_activations(s))[kittens::warpid()];

            kittens::warp::load(activations_smem, g.*InputActivationsPtr,
                       coord<>{inst.start_reduction_col +
                               kittens::warpid() * pipeline::REDUCTION_DIM_PER_WARP});
            kittens::warp::sync();

            rv_t activations_vec;
            kittens::warp::load(activations_vec, activations_smem);
            kittens::warp::sync();

            s.warp_finish_page(pipeline::get_activation_page(s), 1);

            pipeline::consumer_loop(s, g, activations_vec);
        }
    };
    struct storer {
        // Uses 4 full pages for outputs.
        static __device__ void run(const globals &g, megakernel::state<Config> &s) {
            pipeline::storer_loop(s, g);
            kittens::warp::sync();

            if (kittens::laneid() == 0) {
                s.record(megakernel::TEVENT_AT_GMEM_STORE);
                parsed_instruction inst{s};

                kittens::tma::store_async_wait(); // not just read wait! full wait! must
                                         // be visible in global!

                // asm volatile("fence.acq_rel.gpu;\n"); // possible we need sc
                // here but I don't think so.
                atomicAdd(&g.Bar[{inst.layer, opcode - 1, 0}], inst.iters);
                s.record(megakernel::TEVENT_DONE_GMEM_STORE);
            }
        }
    };
};

template <typename Config, typename Globals>
struct downproj : MatVecAddOp<llama_1b_globals::hidden_dim /
                                  llama_1b_globals::matvec_block_size,
                              &Globals::down_weights, &Globals::silu_out,
                              &Globals::hidden_states, OPCODE_DownProjResidual,
                              OPCODE_DownProjResidual - 1, Config, Globals> {};

template <typename Config, typename Globals>
struct o_proj : MatVecAddOp<llama_1b_globals::num_attention_heads,
                            &Globals::o_weights, &Globals::attn_out,
                            &Globals::hidden_states, OPCODE_O_ProjResidual,
                            OPCODE_O_ProjResidual - 1, Config, Globals> {};

