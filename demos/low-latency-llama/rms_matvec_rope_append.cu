#include "hip/hip_runtime.h"
#include "llama.cuh"
#include "utils.cuh"
#include "matvec_pipeline.cuh"

using namespace kittens;
using namespace megakernel;

using globals = llama_1b_globals;

template <typename Config, typename Globals> struct rms_qkv_rope_append {
    static constexpr int opcode =
        OPCODE_RMS_QKV_MatVecRopeAppend; // Op index within the layer --
                                         // controls which barrier to listen to.

    static constexpr int K_BLK_START = 2048 / Globals::matvec_block_size;
    static constexpr int V_BLK_START = 2560 / Globals::matvec_block_size;
    static constexpr int EXPECTED_ARRIVAL_COUNT = 512;

    using rope_t = kittens::sv_fl<Globals::head_dim>;

    __device__ static inline uint8_t *get_rope_cos_ptr(megakernel::state<Config> &s) {
        return (uint8_t *)s.scratch() + Config::SCRATCH_BYTES - 512;
    }
    __device__ static inline uint8_t *get_rope_sin_ptr(megakernel::state<Config> &s) {
        return (uint8_t *)s.scratch() + Config::SCRATCH_BYTES - 256;
    }
    __device__ static inline rope_t &get_rope_cos(megakernel::state<Config> &s) {
        return *reinterpret_cast<rope_t *>(get_rope_cos_ptr(s));
    }
    __device__ static inline rope_t &get_rope_sin(megakernel::state<Config> &s) {
        return *reinterpret_cast<rope_t *>(get_rope_sin_ptr(s));
    }

    struct parsed_instruction {
        int layer_idx, start_block_idx, end_block_idx, iters;
        __device__ inline parsed_instruction(
            typename Config::instruction_t &instruction) {
            layer_idx = instruction[1];       // in units of 1
            start_block_idx = instruction[2]; // in units of 16 elements
            end_block_idx = instruction[3];   // in units of 16 elements
            iters = end_block_idx - start_block_idx;
        }
        __device__ inline parsed_instruction(megakernel::state<Config> &s)
            : parsed_instruction(s.instruction()) {}
    };

    struct pipeline_specifics {

        static __device__ inline void gmem_wait(const Globals &g,
                                                megakernel::state<Config> &s) {
            parsed_instruction inst{s};
            if (inst.layer_idx > 0) {
                while (
                    *(volatile int *)&g.Bar[{inst.layer_idx - 1,
                                             OPCODE_DownProjResidual - 1, 0}] <
                    EXPECTED_ARRIVAL_COUNT) {
                    __nanosleep(Config::GMEM_SPIN_LOOP_SLEEP_NANOS);
                }
            }
        }

        static __device__ inline void
        load_iter(megakernel::state<Config> &s, const globals &g, parsed_instruction &inst,
                  int iter, int col_idx, kittens::st_bf<16, 512> &weight_chunk,
                  kittens::semaphore &sem) {
            auto block_idx = inst.start_block_idx + iter;
            kittens::tma::load_async<dim::ROW, cache_policy::EVICT_FIRST>(
                weight_chunk, g.qkv_weights,
                {inst.layer_idx, block_idx, col_idx}, sem);
        }

        static __device__ inline void store(megakernel::state<Config> &s, const Globals &g,
                                            parsed_instruction &inst,
                                            int output_idx, int output_stage) {
            int block_idx = inst.start_block_idx + output_idx;

            // apply rope

            // even for V, we need to cast from float to bf16
            uint8_t *output_scratch_start =
                pipeline::get_output_start(s, output_stage);

            // kittens::sv_fl<16> &qkv_proj_smem = *reinterpret_cast<kittens::sv_fl<16>
            // *>(output_scratch_start);
            kittens::sv_bf<16> &qkv_proj_smem_bf =
                *reinterpret_cast<kittens::sv_bf<16> *>(output_scratch_start);

            kittens::rv_fl<16> qkv_proj, rope_cos, rope_sin;

            matvec_reduce<Config, kittens::sv_fl<16>, kittens::rv_fl<16>,
                          pipeline::SCRATCH_BYTES_PER_WARP>(
                output_scratch_start, qkv_proj);

            kittens::wait(rope_arrived(s), 0);

            auto head_chunk = block_idx % 4;

            kittens::sv_fl<16> &rope_cos_sv = *reinterpret_cast<kittens::sv_fl<16> *>(
                get_rope_cos_ptr(s) + head_chunk * 64);
            kittens::sv_fl<16> &rope_sin_sv = *reinterpret_cast<kittens::sv_fl<16> *>(
                get_rope_sin_ptr(s) + head_chunk * 64);

            kittens::warp::load(rope_cos, rope_cos_sv);
            kittens::warp::load(rope_sin, rope_sin_sv);

            if (block_idx < V_BLK_START) { // only Q & K need RoPE

                // Fetch the neighbor values
                int mod = (kittens::laneid() & 0b1) ? -1 : 1; // 1 for even, -1 for odd
                kittens::warp::sync();
                float pair_val =
                    __shfl_sync(MASK_ALL, qkv_proj[0][0], kittens::laneid() + mod);

                // Compute RoPE in-place
                if (kittens::laneid() < 16) {
                    // will clean this up later
                    qkv_proj[0][0] =
                        float(qkv_proj[0][0]) * rope_cos[0][0] +
                        float(-1 * mod) * float(pair_val) * rope_sin[0][0];
                }
            }

            kittens::warp::sync();
            kittens::warp::store(qkv_proj_smem_bf, qkv_proj);
            kittens::warp::sync();

            if (kittens::laneid() == 0) {

                if (block_idx < K_BLK_START) { // Q
                    kittens::tma::store_async<cache_policy::EVICT_LAST>(
                        g.q_post_rope, qkv_proj_smem_bf, {0, 0, 0, block_idx});
                } else if (block_idx < V_BLK_START) { // K
                    int base_index =
                        (block_idx - K_BLK_START) * Globals::matvec_block_size;
                    int head_idx = base_index / Globals::head_dim;
                    int dim_idx = (base_index % Globals::head_dim) /
                                  Globals::matvec_block_size;
                    kittens::tma::store_async<cache_policy::EVICT_LAST>(
                        g.k_cache, qkv_proj_smem_bf,
                        {inst.layer_idx, static_cast<int>(g.pos_id), head_idx,
                         dim_idx});
                } else { // V
                    int base_index =
                        (block_idx - V_BLK_START) * Globals::matvec_block_size;
                    int head_idx = base_index / Globals::head_dim;
                    int dim_idx = (base_index % Globals::head_dim) /
                                  Globals::matvec_block_size;
                    kittens::tma::store_async<cache_policy::EVICT_LAST>(
                        g.v_cache, qkv_proj_smem_bf,
                        {inst.layer_idx, static_cast<int>(g.pos_id), head_idx,
                         dim_idx});
                }

                s.record(megakernel::TEVENT_AT_GMEM_STORE);

                kittens::tma::store_async_wait(); // not just read wait! full wait! must
                                         // be visible in global!
                // asm volatile("fence.acq_rel.gpu;\n"); // possible we need sc
                // here but I don't think so.

                atomicAdd(&g.Bar[{inst.layer_idx, opcode - 1, block_idx / 4}],
                          1);
                s.record(megakernel::TEVENT_DONE_GMEM_STORE);
            }

            kittens::warp::sync();
        }
    };

    using pipeline =
        rms_matvec_pipeline<Config, Globals, parsed_instruction,
                            pipeline_specifics, &Globals::hidden_states,
                            &Globals::attn_norm_weights>;

    __device__ static inline kittens::semaphore &rope_arrived(megakernel::state<Config> &s) {
        return s.semaphores()[pipeline::SEM_COUNT];
    }

    struct controller {
        static __device__ int
        release_lid(const Globals &g,
                    typename Config::instruction_t &instruction, int &query) {
            return pipeline::release_lid(g, instruction, query);
        }
        static __device__ int init_semaphores(const Globals &g,
                                              megakernel::state<Config> &s) {
            pipeline::init_semaphores(s);
            init_semaphore(rope_arrived(s), 1);
            return pipeline::SEM_COUNT + 1;
        }
    };
    struct loader {
        static __device__ void run(const Globals &g, megakernel::state<Config> &s) {
            if (kittens::laneid() == 0) {
                auto &rope_cos = get_rope_cos(s);
                auto &rope_sin = get_rope_sin(s);

                auto &sem = rope_arrived(s);
                kittens::tma::expect(sem, rope_cos, rope_sin);

                kittens::tma::load_async<cache_policy::EVICT_LAST>(
                    rope_cos, g.rope_cos, {0, 0, static_cast<int>(g.pos_id), 0},
                    sem);
                kittens::tma::load_async<cache_policy::EVICT_LAST>(
                    rope_sin, g.rope_sin, {0, 0, static_cast<int>(g.pos_id), 0},
                    sem);
            }

            parsed_instruction inst{s};
            pipeline::loader_loop(s, g, inst.layer_idx);
        }
    };
    struct launcher {
        static __device__ void run(const Globals &g, megakernel::state<Config> &s) {

            parsed_instruction inst{s};
            pipeline::launcher_loop(s, g);
        }
    };
    struct consumer {
        static __device__ void run(const Globals &g, megakernel::state<Config> &s) {
            pipeline::consumer_loop(s, g);
        }
    };
    struct storer {
        // Uses 4 full pages for outputs.
        static __device__ void run(const Globals &g, megakernel::state<Config> &s) {
            pipeline::storer_loop(s, g);
        }
    };
};
